
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <float.h>
#include <math.h>
struct timeval startwtime, endwtime;
double seq_time;

// CPU variable
int N; // number of elements
int D; // dimensions
float sigma = 1.0;
float MS_error = 0.0001; // mean shift error
size_t cols;
size_t rows;

//GPU variables
int *gp_N;
int *gp_D;
float *gp_sigma;
size_t gp_pitch;

//CPU matrices
float *x; // initilal matrix
float *y; // final matrix
float **val_y; // validation final matrix
float *m; // mean shift vectrors
// GPU matrices
float *gp_x;
float *gp_y;
float *gp_y_new;
float *gp_m; // mean shift vectrors
float *gp_g; // gaussian
float *gp_numer; // numerator
float *gp_denom; // denominator




__global__ void cuda_mean_shift(void);

void init(void);
void mean_shift(void);
void test(void);
void free_arrays(void);


int main(int argc, char **argv) {


  if (argc != 3) {
    printf("Usage: %s N data \n D dimension  )\n",
           argv[0]);
    exit(1);
  }
  


  N =  atoi(argv[1]);
  D =  atoi(argv[2]);
  
  init();

  gettimeofday (&startwtime, NULL);
  
  mean_shift();

  gettimeofday (&endwtime, NULL);
  
  seq_time = (long double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6 + endwtime.tv_sec - startwtime.tv_sec);

  printf("KNN wall clock time = %f\n", seq_time);
  

  // Copy matrices to memory
  hipMemcpy(y,gp_y, N*D*sizeof(float), hipMemcpyDeviceToHost);
  test();

  free_arrays();
}




/** procedure init() : initialize array "a" with data **/
void init() {
  int i,j;
  int ret_code=0;
  
  FILE *f;

  // Allocate system memmory
/*
  x = (float **) malloc(N * sizeof(float*));
  for (i = 0; i < N; i++) {
    x[i] = (float *) malloc(D * sizeof(float));
  }

  y = (float **) malloc(N * sizeof(float*));
  for (i = 0; i < N; i++) {
    y[i] = (float *) malloc(D * sizeof(float));
  }


  m = (float **) malloc(N * sizeof(float*));
  for (i = 0; i < N; i++) {
    m[i] = (float *) malloc(D * sizeof(float));
  }
  
*/

//(x)[cols] =malloc(sizeof(*x) * rows);
//(y)[cols] = malloc(sizeof(*y) *rows);
//(m)[cols] = malloc(sizeof(*m) *rows);
x = (float *) malloc(N*D * sizeof(float));
y = (float *) malloc(N*D * sizeof(float));
m = (float *) malloc(N*D * sizeof(float));


  val_y = (float **) malloc(N * sizeof(float*));
  for (i = 0; i < N; i++) {
    val_y[i] = (float *) malloc(D * sizeof(float));
  }
  // Allocate GPU variables

  hipMalloc( (void**)&gp_N , sizeof(int));
  hipMalloc( (void**)&gp_D , sizeof(int));
  hipMalloc( (void**)&gp_sigma , sizeof(float));


  // Allocate GPU matrices

  hipMallocPitch(&gp_x, &gp_pitch, N * sizeof(float), D);
  hipMallocPitch(&gp_y, &gp_pitch, N * sizeof(float), D);
  hipMallocPitch(&gp_y_new, &gp_pitch, N * sizeof(float), D);
  hipMallocPitch(&gp_m, &gp_pitch, N * sizeof(float), D);
  hipMallocPitch(&gp_g, &gp_pitch, N * sizeof(float), N);
  hipMallocPitch(&gp_numer, &gp_pitch, N * sizeof(float), D);
  hipMallocPitch(&gp_denom, &gp_pitch, N * sizeof(float), D);


  f = fopen("../data/r15.txt","r");
  for (i = 0; i < N; ++i) {
    for (j = 0; j < D; ++j) {
      ret_code = fscanf(f, "%f\t", &x[i*D+j]);
    }
    ret_code = fscanf(f,"\n");   
  }
  fclose(f);

  for (i = 0; i < N; ++i) {
    for (j = 0; j < D; ++j) {
      y[i*D+j] = x[i*D+j];
    }
  }
  
  for (i = 0; i < N; ++i) {
    for (j = 0; j < D; ++j) {
      m[i*D+j] =FLT_MAX;
    }
  }

  // Copy variables to GPU
  hipMemcpy(gp_N,&N,sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(gp_D,&D,sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(gp_sigma,&sigma,sizeof(int),hipMemcpyHostToDevice);

  // Copy matrices to GPU
  hipMemcpy(gp_x,x,N*D*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gp_y,y,N*D*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gp_m,m,N*D*sizeof(float), hipMemcpyHostToDevice);

}





__global__ void cuda_mean_shift(int *N,int *D,float *sigma,float *x, float *y, float *y_new,float *m, float *g,float *numer,float *denom,size_t pitch){

 int tid = threadIdx.x;
 //int tid = blockIdx.x;
 int j,z;
 float dist=0; 
    //for(i=0;i<N;i++){
      
      for(j=0;j<*N;j++){
        float *row_g = (float *)((char*)g + j * pitch);
        for(z=0;z<*D;z++){
          dist += powf((y[tid*(*D)+z]-x[j*(*D)+z]),2.0);

        }
        dist = sqrtf(dist);

        if (dist>powf(*sigma,2.0)) row_g[tid]=0;
        else row_g[tid] = expf(-dist/(2*powf(*sigma,2.0)));

        if (tid==j) row_g[tid] += 1;
        dist=0;

        for(z=0;z<*D;z++){
          float *row_numer = (float *)((char*)numer + z * pitch);
          float *row_denom = (float *)((char*)denom + z * pitch);
          row_numer[tid] += row_g[tid]*x[j*(*D)+z]; 
          row_denom[tid] +=row_g[tid];
        }

      }

      for(z=0;z<*D;z++){
        float *row_y_new = (float *)((char*)y_new + z * pitch);
        float *row_numer = (float *)((char*)numer + z * pitch);
        float *row_denom = (float *)((char*)denom + z * pitch);
        row_y_new[tid] = row_numer[tid]/row_denom[tid];
        m[tid*(*D)+z] = row_y_new[tid] - y[tid*(*D)+z];
      }
    
      for(z=0;z<*D;z++){
        float *row_y_new = (float *)((char*)y_new + z * pitch);
        float *row_numer = (float *)((char*)numer + z * pitch);
        float *row_denom = (float *)((char*)denom + z * pitch);
        y[tid*(*D)+z] = row_y_new[tid];
        row_numer[tid] = 0;
        row_denom[tid] = 0;
      }


   // }


}





void mean_shift() {
  int iter = 0;
  int i,z;
  //float dist = 0;
  float er = FLT_MAX;
  float temp = 0;


  

  while(er > MS_error) {
     
    iter++;
    er = 0;
    
    cuda_mean_shift<<<1,N>>>(gp_N,gp_D,gp_sigma,gp_x,gp_y,gp_y_new,gp_m,gp_g,gp_numer,gp_denom,gp_pitch); 
   hipMemcpy(m,gp_m,N*D*sizeof(float), hipMemcpyDeviceToHost); 
   for(i=0;i<N;i++){
     for(z=0;z<D;z++){
       temp += pow(m[i*D+z],2);
     }
     er += temp;
     temp = 0;
   }
   
   er = sqrt(er);

   //printf("%lf,,,,,,%lf\n",y_new[1][1],y[1][1]); 

  //  printf("Iteration = %d, Error = %lf\n",iter,er);

  } 
  

}







void test() {
    int i,j;
    int pass = 1;
    int ret_code = 0;
    int count = 0;
    float error = 0.0001; 
    FILE *f;

    f = fopen("../data/validation_r15.txt","r");
    for (i = 0; i < N; ++i) {
      for (j = 0; j < D; ++j) {
        ret_code = fscanf(f, "%f\t", &val_y[i][j]);
      }
        ret_code = fscanf(f,"\n");
    }
    fclose(f);
    //printf("%f\n",fabs(val_y[1][1]-y[1][1]));
    
   f = fopen("yout.txt","w+");
   for (i = 0; i < N; i++) {
      for (j = 0; j < D; j++) {
        fprintf(f,"%f\t",y[i*D+j]);
      }
      fprintf(f,"\n");
    }
    fclose(f);



    for (i = 0; i < N; ++i) {
      for (j = 0; j < D; ++j) {

	//pass &= (abs(val_y[i][j] - y[i][j]) <= error);
        
	if(fabs(val_y[i][j] - y[i*D+j]) > error){
	  count++;
          pass=0;
        }
      }
    }
    printf(" TEST %s\n",(pass) ? "PASSed" : "FAILed");
    printf("%d\n",count);



}


void free_arrays(){

free(x);
free(y);
free(val_y);
free(m);

hipFree(gp_N); 
hipFree(gp_D);
hipFree(gp_sigma);
hipFree(gp_x);
hipFree(gp_y);
hipFree(gp_y_new);
hipFree(gp_m);
hipFree(gp_g);
hipFree(gp_numer);
hipFree(gp_denom);

}



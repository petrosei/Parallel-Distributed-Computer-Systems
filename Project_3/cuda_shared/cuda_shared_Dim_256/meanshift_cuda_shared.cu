
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <float.h>
#include <math.h>
struct timeval startwtime, endwtime;
double seq_time;

// CPU variable
int N = 256; // number of elements
int D = 64; // dimensions
int nT = 64; // number of threads per block
float sigma = 10.0;
float MS_error = 0.0001; // mean shift error
float error = 127*0.1; // mean value of dataset * 10^(-3) 
int iter_limit = 4;
size_t cols;
size_t rows;

//GPU variables
int *gp_N;
int *gp_D;
float *gp_sigma;
size_t gp_pitch;

//CPU matrices
float *x; // initilal matrix
float *y; // final matrix
float **val_y; // validation final matrix
float *m; // mean shift vectrors
// GPU matrices
float *gp_x;
float *gp_y;
float *gp_y_new;
float *gp_m; // mean shift vectrors
float *gp_g; // gaussian
float *gp_numer; // numerator
float *gp_denom; // denominator




__global__ void cuda_mean_shift(void);

void init(void);
void mean_shift(void);
void test(void);
void free_arrays(void);


int main(int argc, char **argv) {

  MS_error = MS_error*sigma;  

  init();

  gettimeofday (&startwtime, NULL);
  
  mean_shift();

  gettimeofday (&endwtime, NULL);
  
  seq_time = (long double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6 + endwtime.tv_sec - startwtime.tv_sec);

  printf("KNN wall clock time = %f\n", seq_time);
  

  // Copy matrices to memory
  hipMemcpy(y,gp_y, N*D*sizeof(float), hipMemcpyDeviceToHost);
  test();

  free_arrays();
}




/** procedure init() : initialize array "a" with data **/
void init() {
  int i,j;
  int ret_code=0;
  
  FILE *f;

  // Allocate system memmory

  x = (float *) malloc(N*D * sizeof(float));
  y = (float *) malloc(N*D * sizeof(float));
  m = (float *) malloc(N*D * sizeof(float));


  val_y = (float **) malloc(N * sizeof(float*));
  for (i = 0; i < N; i++) {
    val_y[i] = (float *) malloc(D * sizeof(float));
  }
  // Allocate GPU variables

  hipMalloc( (void**)&gp_N , sizeof(int));
  hipMalloc( (void**)&gp_D , sizeof(int));
  hipMalloc( (void**)&gp_sigma , sizeof(float));


  // Allocate GPU matrices

  hipMallocPitch(&gp_x, &gp_pitch, N * sizeof(float), D);
  hipMallocPitch(&gp_y, &gp_pitch, N * sizeof(float), D);
  hipMallocPitch(&gp_y_new, &gp_pitch, N * sizeof(float), D);
  hipMallocPitch(&gp_m, &gp_pitch, N * sizeof(float), D);
  hipMallocPitch(&gp_g, &gp_pitch, N * sizeof(float), N);
  hipMallocPitch(&gp_numer, &gp_pitch, N * sizeof(float), D);
  hipMallocPitch(&gp_denom, &gp_pitch, N * sizeof(float), D);


  f = fopen("../../data/Dim/Dim_256x64.txt","r");
  for (i = 0; i < N; ++i) {
    for (j = 0; j < D; ++j) {
      ret_code = fscanf(f, "%f\t", &x[i*D+j]);
    }
    ret_code = fscanf(f,"\n");   
  }
  fclose(f);

  for (i = 0; i < N; ++i) {
    for (j = 0; j < D; ++j) {
      y[i*D+j] = x[i*D+j];
    }
  }
  
  for (i = 0; i < N; ++i) {
    for (j = 0; j < D; ++j) {
      m[i*D+j] =FLT_MAX;
    }
  }

  // Copy variables to GPU
  hipMemcpy(gp_N,&N,sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(gp_D,&D,sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(gp_sigma,&sigma,sizeof(int),hipMemcpyHostToDevice);

  // Copy matrices to GPU
  hipMemcpy(gp_x,x,N*D*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gp_y,y,N*D*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gp_m,m,N*D*sizeof(float), hipMemcpyHostToDevice);

}





__global__ void cuda_mean_shift(int *N,int *D,float *sigma,float *x, float *y, float *y_new,float *m, float *g,float *numer,float *denom,size_t pitch){

 int tids = threadIdx.x;
 int tid = threadIdx.x + blockIdx.x * blockDim.x;
 int j,z;
 float r;
 float dist=0;
 __shared__ float s_y[64];
 __shared__ float s_m[64]; 
__shared__ float s_y_new[64];
__shared__ float s_numer[64];   
__shared__ float s_denom[64];
__shared__ float diff[64];

      s_y[tids] = y[tid];  
      s_m[tids] = m[tid];
      s_numer[tids] = 0;
      s_denom[tids] = 0;
    __syncthreads(); 
    if(tid<*N*(*D)){
      
      for(j=0;j<*N;j++){
        float *row_g = (float *)((char*)g + j * pitch);
          diff[tids] = powf((s_y[tids]-x[j*(*D)+tids]),2.0);
        __syncthreads();
        for(z=0;z<*D;z++){
          dist += diff[z];
        }
        
        dist = sqrtf(dist);

        if (dist>powf(*sigma,2.0)) row_g[(tid-tids)/blockDim.x]=0;
        else row_g[(tid-tids)/blockDim.x] = expf(-dist/(2*powf(*sigma,2.0)));

        if ((tid-tids)/blockDim.x == j) row_g[(tid-tids)/blockDim.x] += 1;
        dist=0;
          s_numer[tids] += row_g[(tid-tids)/blockDim.x]*x[j*(*D)+tids]; 
          s_denom[tids] +=row_g[(tid-tids)/blockDim.x];
        __syncthreads();

      }

        s_y_new[tids] = s_numer[tids]/s_denom[tids];
        s_m[tids] = s_y_new[tids] - s_y[tids];
    
        s_y[tids] = s_y_new[tids];
        s_numer[tids] = 0;
        s_denom[tids] = 0;
__syncthreads();

    }
    
      y[tid] = s_y[tids];
      m[tid] = s_m[tids];
    __syncthreads();


}





void mean_shift() {
  int iter = 0;
  int i,z;
  float er = FLT_MAX;
  float last_er = FLT_MAX;
  float temp = 0;


  

  while(er > MS_error && iter<iter_limit && last_er >= er) {
     
    last_er = er;
    iter++;
    er = 0;
    
    cuda_mean_shift<<<N,nT>>>(gp_N,gp_D,gp_sigma,gp_x,gp_y,gp_y_new,gp_m,gp_g,gp_numer,gp_denom,gp_pitch); 
   hipMemcpy(m,gp_m,N*D*sizeof(float), hipMemcpyDeviceToHost); 
   for(i=0;i<N;i++){
     for(z=0;z<D;z++){
       temp += pow(m[i*D+z],2);
     }
     er += temp;
     temp = 0;
   }
   
   er = sqrt(er);


    printf("Iteration = %d, Error = %lf\n",iter,er);

  } 
  

}







void test() {
    int i,j;
    int pass = 1;
    int ret_code = 0;
    int count = 0;
    FILE *f;

    f = fopen("../../data/Dim/validation_Dim_256x64.txt","r");
    for (i = 0; i < N; ++i) {
      for (j = 0; j < D; ++j) {
        ret_code = fscanf(f, "%f\t", &val_y[i][j]);
      }
        ret_code = fscanf(f,"\n");
    }
    fclose(f);
    
   f = fopen("yout.txt","w+");
   for (i = 0; i < N; i++) {
      for (j = 0; j < D; j++) {
        fprintf(f,"%f\t",y[i*D+j]);
      }
      fprintf(f,"\n");
    }
    fclose(f);



    for (i = 0; i < N; ++i) {
      for (j = 0; j < D; ++j) {

        
	if(fabs(val_y[i][j] - y[i*D+j]) > error){
	  count++;
          pass=0;
        }
      }
    }
    printf(" TEST %s\n",(pass) ? "PASSed" : "FAILed");
    printf(" Errors = %d\n",count);



}


void free_arrays(){

free(x);
free(y);
free(val_y);
free(m);

hipFree(gp_N); 
hipFree(gp_D);
hipFree(gp_sigma);
hipFree(gp_x);
hipFree(gp_y);
hipFree(gp_y_new);
hipFree(gp_m);
hipFree(gp_g);
hipFree(gp_numer);
hipFree(gp_denom);

}



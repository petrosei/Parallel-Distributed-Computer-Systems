
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <float.h>
#include <math.h>
struct timeval startwtime, endwtime;
double seq_time;

// CPU variable
int N = 5000; // number of elements
int D = 2; // dimensions
int nT = 500; // number of threads per block
float sigma = 250.0;
float MS_error = 0.0001; // mean shift error
float error = 504820*0.001; // mean value of dataset * 10^(-3) 
int iter_limit = 30;
size_t cols;
size_t rows;

//GPU variables
int *gp_N;
int *gp_D;
float *gp_sigma;
size_t gp_pitch;

//CPU matrices
float *x; // initilal matrix
float *y; // final matrix
float **val_y; // validation final matrix
float *m; // mean shift vectrors
// GPU matrices
float *gp_x;
float *gp_y;
float *gp_y_new;
float *gp_m; // mean shift vectrors
float *gp_g; // gaussian
float *gp_numer; // numerator
float *gp_denom; // denominator




__global__ void cuda_mean_shift(void);

void init(void);
void mean_shift(void);
void test(void);
void free_arrays(void);


int main(int argc, char **argv) {

  MS_error = MS_error*sigma;  

  init();

  gettimeofday (&startwtime, NULL);
  
  mean_shift();

  gettimeofday (&endwtime, NULL);
  
  seq_time = (long double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6 + endwtime.tv_sec - startwtime.tv_sec);

  printf("KNN wall clock time = %f\n", seq_time);
  

  // Copy matrices to memory
  hipMemcpy(y,gp_y, N*D*sizeof(float), hipMemcpyDeviceToHost);
  test();

  free_arrays();
}




/** procedure init() : initialize array "a" with data **/
void init() {
  int i,j;
  int ret_code=0;
  
  FILE *f;

  // Allocate system memmory

  x = (float *) malloc(N*D * sizeof(float));
  y = (float *) malloc(N*D * sizeof(float));
  m = (float *) malloc(N*D * sizeof(float));


  val_y = (float **) malloc(N * sizeof(float*));
  for (i = 0; i < N; i++) {
    val_y[i] = (float *) malloc(D * sizeof(float));
  }
  // Allocate GPU variables

  hipMalloc( (void**)&gp_N , sizeof(int));
  hipMalloc( (void**)&gp_D , sizeof(int));
  hipMalloc( (void**)&gp_sigma , sizeof(float));


  // Allocate GPU matrices

  hipMallocPitch(&gp_x, &gp_pitch, N * sizeof(float), D);
  hipMallocPitch(&gp_y, &gp_pitch, N * sizeof(float), D);
  hipMallocPitch(&gp_y_new, &gp_pitch, N * sizeof(float), D);
  hipMallocPitch(&gp_m, &gp_pitch, N * sizeof(float), D);
  hipMallocPitch(&gp_g, &gp_pitch, N * sizeof(float), N);
  hipMallocPitch(&gp_numer, &gp_pitch, N * sizeof(float), D);
  hipMallocPitch(&gp_denom, &gp_pitch, N * sizeof(float), D);


  f = fopen("../../data/S_set/S_set_5000x2.txt","r");
  for (i = 0; i < N; ++i) {
    for (j = 0; j < D; ++j) {
      ret_code = fscanf(f, "%f\t", &x[i*D+j]);
    }
    ret_code = fscanf(f,"\n");   
  }
  fclose(f);

  for (i = 0; i < N; ++i) {
    for (j = 0; j < D; ++j) {
      y[i*D+j] = x[i*D+j];
    }
  }
  
  for (i = 0; i < N; ++i) {
    for (j = 0; j < D; ++j) {
      m[i*D+j] =FLT_MAX;
    }
  }

  // Copy variables to GPU
  hipMemcpy(gp_N,&N,sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(gp_D,&D,sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(gp_sigma,&sigma,sizeof(int),hipMemcpyHostToDevice);

  // Copy matrices to GPU
  hipMemcpy(gp_x,x,N*D*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gp_y,y,N*D*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gp_m,m,N*D*sizeof(float), hipMemcpyHostToDevice);

}





__global__ void cuda_mean_shift(int *N,int *D,float *sigma,float *x, float *y, float *y_new,float *m, float *g,float *numer,float *denom,size_t pitch){

 int tids = threadIdx.x;
 int tid = threadIdx.x + blockIdx.x * blockDim.x;
 int j,z;
 float r;
 float dist=0;
 __shared__ float s_y[500*2];
 __shared__ float s_m[500*2]; 
__shared__ float s_y_new[500*2];
__shared__ float s_numer[500*2];   
__shared__ float s_denom[500*2];

    for(z=0;z<*D;z++){
      s_y[tids*(*D)+z] = y[tid*(*D)+z];  
      s_m[tids*(*D)+z] = m[tid*(*D)+z];
      s_numer[tids*(*D)+z] = 0;
      s_denom[tids*(*D)+z] = 0;
    }
    __syncthreads(); 
    if(tid<*N){
      
      for(j=0;j<*N;j++){
        float *row_g = (float *)((char*)g + j * pitch);
        for(z=0;z<*D;z++){
          dist += powf((s_y[tids*(*D)+z]-x[j*(*D)+z]),2.0);

        }
        dist = sqrtf(dist);

        if (dist>powf(*sigma,2.0)) row_g[tid]=0;
        else row_g[tid] = expf(-dist/(2*powf(*sigma,2.0)));

        if (tid==j) row_g[tid] += 1;
        dist=0;
        for(z=0;z<*D;z++){
          s_numer[tids*(*D)+z] += row_g[tid]*x[j*(*D)+z]; 
          s_denom[tids*(*D)+z] +=row_g[tid];
        }
        __syncthreads();

      }

      for(z=0;z<*D;z++){
        s_y_new[tids*(*D)+z] = s_numer[tids*(*D)+z]/s_denom[tids*(*D)+z];
        s_m[tids*(*D)+z] = s_y_new[tids*(*D)+z] - s_y[tids*(*D)+z];
      }
    
      for(z=0;z<*D;z++){
        s_y[tids*(*D)+z] = s_y_new[tids*(*D)+z];
        s_numer[tids*(*D)+z] = 0;
        s_denom[tids*(*D)+z] = 0;
      }
__syncthreads();

    }
    
    for(z=0;z<*D;z++){
      y[tid*(*D)+z] = s_y[tids*(*D)+z];
      m[tid*(*D)+z] = s_m[tids*(*D)+z];
    }
    __syncthreads();


}





void mean_shift() {
  int iter = 0;
  int i,z;
  float er = FLT_MAX;
  float last_er = FLT_MAX;
  float temp = 0;


  

  while(er > MS_error && iter<iter_limit && last_er >= er) {
     
    last_er = er;
    iter++;
    er = 0;
    
    cuda_mean_shift<<<N/nT,nT>>>(gp_N,gp_D,gp_sigma,gp_x,gp_y,gp_y_new,gp_m,gp_g,gp_numer,gp_denom,gp_pitch); 
   hipMemcpy(m,gp_m,N*D*sizeof(float), hipMemcpyDeviceToHost); 
   for(i=0;i<N;i++){
     for(z=0;z<D;z++){
       temp += pow(m[i*D+z],2);
     }
     er += temp;
     temp = 0;
   }
   
   er = sqrt(er);


    printf("Iteration = %d, Error = %lf\n",iter,er);

  } 
  

}







void test() {
    int i,j;
    int pass = 1;
    int ret_code = 0;
    int count = 0;
    FILE *f;

    f = fopen("../../data/S_set/validation_S_set_5000x2.txt","r");
    for (i = 0; i < N; ++i) {
      for (j = 0; j < D; ++j) {
        ret_code = fscanf(f, "%f\t", &val_y[i][j]);
      }
        ret_code = fscanf(f,"\n");
    }
    fclose(f);
    
   f = fopen("yout.txt","w+");
   for (i = 0; i < N; i++) {
      for (j = 0; j < D; j++) {
        fprintf(f,"%f\t",y[i*D+j]);
      }
      fprintf(f,"\n");
    }
    fclose(f);



    for (i = 0; i < N; ++i) {
      for (j = 0; j < D; ++j) {

        
	if(fabs(val_y[i][j] - y[i*D+j]) > error){
	  count++;
          pass=0;
        }
      }
    }
    printf(" TEST %s\n",(pass) ? "PASSed" : "FAILed");
    printf(" Errors = %d\n",count);



}


void free_arrays(){

free(x);
free(y);
free(val_y);
free(m);

hipFree(gp_N); 
hipFree(gp_D);
hipFree(gp_sigma);
hipFree(gp_x);
hipFree(gp_y);
hipFree(gp_y_new);
hipFree(gp_m);
hipFree(gp_g);
hipFree(gp_numer);
hipFree(gp_denom);

}


